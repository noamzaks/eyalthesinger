#include "hip/hip_runtime.h"
/*
 * sha256.cu Implementation of SHA256 Hashing
 *
 * Date: 12 June 2019
 * Revision: 1
 * *
 * Based on the public domain Reference Implementation in C, by
 * Brad Conte, original code here:
 *
 * https://github.com/B-Con/crypto-algorithms
 *
 * This file is released into the Public Domain.
 */

/*************************** HEADER FILES ***************************/
#include <cassert>
#include <hip/hip_runtime_api.h>
#include <memory.h>
#include <stdlib.h>

extern "C" {
#include "sha256.cuh"
}
/****************************** MACROS ******************************/
#define SHA256_BLOCK_SIZE 32 // SHA256 outputs a 32 byte digest

/**************************** DATA TYPES ****************************/

typedef struct {
  BYTE data[64];
  WORD datalen;
  unsigned long long bitlen;
  WORD state[8];
} CUDA_SHA256_CTX;

/****************************** MACROS ******************************/
#ifndef ROTLEFT
#define ROTLEFT(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
#endif

#define ROTRIGHT(a, b) (((a) >> (b)) | ((a) << (32 - (b))))

#define CH(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x, 2) ^ ROTRIGHT(x, 13) ^ ROTRIGHT(x, 22))
#define EP1(x) (ROTRIGHT(x, 6) ^ ROTRIGHT(x, 11) ^ ROTRIGHT(x, 25))
#define SIG0(x) (ROTRIGHT(x, 7) ^ ROTRIGHT(x, 18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x, 17) ^ ROTRIGHT(x, 19) ^ ((x) >> 10))

/**************************** VARIABLES *****************************/
__constant__ WORD k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1,
    0x923f82a4, 0xab1c5ed5, 0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, 0xe49b69c1, 0xefbe4786,
    0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147,
    0x06ca6351, 0x14292967, 0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, 0xa2bfe8a1, 0xa81a664b,
    0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a,
    0x5b9cca4f, 0x682e6ff3, 0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

/*********************** FUNCTION DEFINITIONS ***********************/
__device__ __forceinline__ void cuda_sha256_transform(CUDA_SHA256_CTX *ctx,
                                                      const BYTE data[]) {
  WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

  for (i = 0, j = 0; i < 16; ++i, j += 4)
    m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) |
           (data[j + 3]);
  for (; i < 64; ++i)
    m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

  a = ctx->state[0];
  b = ctx->state[1];
  c = ctx->state[2];
  d = ctx->state[3];
  e = ctx->state[4];
  f = ctx->state[5];
  g = ctx->state[6];
  h = ctx->state[7];

  for (i = 0; i < 64; ++i) {
    t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
    t2 = EP0(a) + MAJ(a, b, c);
    h = g;
    g = f;
    f = e;
    e = d + t1;
    d = c;
    c = b;
    b = a;
    a = t1 + t2;
  }

  ctx->state[0] += a;
  ctx->state[1] += b;
  ctx->state[2] += c;
  ctx->state[3] += d;
  ctx->state[4] += e;
  ctx->state[5] += f;
  ctx->state[6] += g;
  ctx->state[7] += h;
}

__device__ void cuda_sha256_init(CUDA_SHA256_CTX *ctx) {
  ctx->datalen = 0;
  ctx->bitlen = 0;
  ctx->state[0] = 0x6a09e667;
  ctx->state[1] = 0xbb67ae85;
  ctx->state[2] = 0x3c6ef372;
  ctx->state[3] = 0xa54ff53a;
  ctx->state[4] = 0x510e527f;
  ctx->state[5] = 0x9b05688c;
  ctx->state[6] = 0x1f83d9ab;
  ctx->state[7] = 0x5be0cd19;
}

__device__ void cuda_sha256_update(CUDA_SHA256_CTX *ctx, const BYTE data[],
                                   size_t len) {
  WORD i;

  for (i = 0; i < len; ++i) {
    if (data[i] == 0) {
      break;
    }

    ctx->data[ctx->datalen] = data[i];
    ctx->datalen++;
    if (ctx->datalen == 64) {
      cuda_sha256_transform(ctx, ctx->data);
      ctx->bitlen += 512;
      ctx->datalen = 0;
    }
  }
}

__device__ void cuda_sha256_final(CUDA_SHA256_CTX *ctx, BYTE out[],
                                  BYTE *indata, WORD inlen, BYTE *target_hash) {
  WORD i;

  i = ctx->datalen;

  // Pad whatever data is left in the buffer.
  if (ctx->datalen < 56) {
    ctx->data[i++] = 0x80;
    while (i < 56)
      ctx->data[i++] = 0x00;
  } else {
    ctx->data[i++] = 0x80;
    while (i < 64)
      ctx->data[i++] = 0x00;
    cuda_sha256_transform(ctx, ctx->data);
    memset(ctx->data, 0, 56);
  }

  // Append to the padding the total message's length in bits and transform.
  ctx->bitlen += ctx->datalen * 8;
  ctx->data[63] = ctx->bitlen;
  ctx->data[62] = ctx->bitlen >> 8;
  ctx->data[61] = ctx->bitlen >> 16;
  ctx->data[60] = ctx->bitlen >> 24;
  ctx->data[59] = ctx->bitlen >> 32;
  ctx->data[58] = ctx->bitlen >> 40;
  ctx->data[57] = ctx->bitlen >> 48;
  ctx->data[56] = ctx->bitlen >> 56;
  cuda_sha256_transform(ctx, ctx->data);

  // Since this implementation uses little endian byte ordering and SHA uses big
  // endian, reverse all the bytes when copying the final state to the output
  // hash.
  BYTE hash[32];
  for (i = 0; i < 4; ++i) {
    hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
    hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
  }

  // compare hash to wanted hash

  bool is_equal = true;
  for (int i = 0; i < SHA256_BLOCK_SIZE; i++) {
    if (target_hash[i] != hash[i]) {
      is_equal = false;
      break;
    }
  }

  // if success, copy indata to out
  if (is_equal) {
    for (int i = 0; i < inlen; i++) {
      out[i] = indata[i];
    }
    // hipMemcpy(out, indata, inlen, hipMemcpyDeviceToDevice);
  }
}

__global__ void kernel_sha256_hash(BYTE *indata, WORD inlen, BYTE *outdata,
                                   WORD n_batch, char *result) {
  WORD thread = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread >= n_batch) {
    return;
  }
  BYTE *in = indata + thread * inlen;
  BYTE *out = outdata;
  CUDA_SHA256_CTX ctx;
  cuda_sha256_init(&ctx);
  cuda_sha256_update(&ctx, in, inlen);
  cuda_sha256_final(&ctx, out, indata, inlen, (unsigned char *)result);
}

extern "C" {
void mcm_cuda_sha256_hash_batch(BYTE *in, WORD inlen, BYTE *out, WORD n_batch,
                                char result[SHA256_BLOCK_SIZE]) {
  BYTE *cuda_indata;
  BYTE *cuda_outdata;
  char *cuda_result;
  hipMalloc(&cuda_indata, inlen * n_batch);
  hipMalloc(&cuda_outdata, SHA256_BLOCK_SIZE);
  hipMemcpy(cuda_indata, in, inlen * n_batch, hipMemcpyHostToDevice);
  hipMalloc(&cuda_result, SHA256_BLOCK_SIZE);
  hipMemcpy(cuda_result, result, SHA256_BLOCK_SIZE, hipMemcpyHostToDevice);

  WORD thread = 256;
  WORD block = (n_batch + thread - 1) / thread;

  kernel_sha256_hash<<<block, thread>>>(cuda_indata, inlen, cuda_outdata,
                                        n_batch, cuda_result);
  hipMemcpy(out, cuda_outdata, SHA256_BLOCK_SIZE, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("Error cuda sha256 hash: %s \n", hipGetErrorString(error));
  }
  hipFree(cuda_indata);
  hipFree(cuda_outdata);
  hipFree(cuda_result);
}
}

// Maps 0..9 and a..f to 0..15
static char hex_values[256] = {
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 1, 2, 3, 4, 5,  6,  7,  8,  9,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 10, 11, 12, 13, 14, 15, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,
};

static void load_hex(const char *hexdump, char *data) {
  while (*hexdump != '\0') {
    assert(('0' <= *hexdump && *hexdump <= '9') ||
           ('a' <= *hexdump && *hexdump <= 'f'));
    assert(('0' <= *(hexdump + 1) && *(hexdump + 1) <= '9') ||
           ('a' <= *(hexdump + 1) && *(hexdump + 1) <= 'f'));

    *data = hex_values[*hexdump] * 16 + hex_values[*(hexdump + 1)];

    data++;
    hexdump += 2;
  }
}

#define MAX_PASSWORD_LENGTH 40

int main(int argc, char **argv) {
  char *wordlist;
  FILE *file = fopen(argv[1], "r");
  assert(file != NULL);

  // Get the size of the file
  fseek(file, 0, SEEK_END);
  long file_size = ftell(file);
  rewind(file);

  wordlist = (char *)malloc(file_size * sizeof(char));
  assert(wordlist != NULL);

  size_t bytes_read = fread(wordlist, sizeof(char), file_size, file);

  fclose(file);

  char output[MAX_PASSWORD_LENGTH + 1 + SHA256_BLOCK_SIZE];
  output[0] = '\0';

  assert(strlen(argv[2]) == SHA256_BLOCK_SIZE * 2);
  char result[SHA256_BLOCK_SIZE];
  load_hex(argv[2], result);

  int n_batch = bytes_read / (MAX_PASSWORD_LENGTH + 1);
  printf("Batch count: %d\n", n_batch);

  mcm_cuda_sha256_hash_batch((unsigned char *)wordlist, 40,
                             (unsigned char *)output, n_batch, result);

  printf("The password is: %s\n", output);

  return 0;
}