
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_LINE_LENGTH 1024
#define SHA256_LENGTH 32

// Maps 0..9 and a..f to 0..15
static char hex_values[256] = {
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 1, 2, 3, 4, 5,  6,  7,  8,  9,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 10, 11, 12, 13, 14, 15, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,  0,  0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0,  0,  0,  0,  0,
};

static void load_hex(const char *hexdump, char *data) {
  while (*hexdump != '\0') {
    assert(('0' <= *hexdump && *hexdump <= '9') ||
           ('a' <= *hexdump && *hexdump <= 'f'));
    assert(('0' <= *(hexdump + 1) && *(hexdump + 1) <= '9') ||
           ('a' <= *(hexdump + 1) && *(hexdump + 1) <= 'f'));

    *data = hex_values[*hexdump] * 16 + hex_values[*(hexdump + 1)];

    data++;
    hexdump += 2;
  }
}

#define ROTLEFT(a, b) (((a) << (b)) | ((a) >> (32 - (b))))
#define ROTRIGHT(a, b) (((a) >> (b)) | ((a) << (32 - (b))))

#define CH(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x, 2) ^ ROTRIGHT(x, 13) ^ ROTRIGHT(x, 22))
#define EP1(x) (ROTRIGHT(x, 6) ^ ROTRIGHT(x, 11) ^ ROTRIGHT(x, 25))
#define SIG0(x) (ROTRIGHT(x, 7) ^ ROTRIGHT(x, 18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x, 17) ^ ROTRIGHT(x, 19) ^ ((x) >> 10))

__device__ static const uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1,
    0x923f82a4, 0xab1c5ed5, 0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
    0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, 0xe49b69c1, 0xefbe4786,
    0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147,
    0x06ca6351, 0x14292967, 0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
    0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, 0xa2bfe8a1, 0xa81a664b,
    0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a,
    0x5b9cca4f, 0x682e6ff3, 0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
    0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__device__ void sha256_transform(uint32_t state[8], uint8_t data[64]) {
  uint32_t a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

  for (i = 0, j = 0; i < 16; ++i, j += 4)
    m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) |
           (data[j + 3]);
  for (; i < 64; ++i)
    m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

  a = state[0];
  b = state[1];
  c = state[2];
  d = state[3];
  e = state[4];
  f = state[5];
  g = state[6];
  h = state[7];

  for (i = 0; i < 64; ++i) {
    t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
    t2 = EP0(a) + MAJ(a, b, c);
    h = g;
    g = f;
    f = e;
    e = d + t1;
    d = c;
    c = b;
    b = a;
    a = t1 + t2;
  }

  state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
  state[4] += e;
  state[5] += f;
  state[6] += g;
  state[7] += h;
}

__device__ void sha256(const char *data, char result[SHA256_LENGTH]) {
  uint8_t current_data[64];
  int length = 0;
  uint64_t bit_length = 0;
  uint32_t state[8] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
                       0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};

  while (*data != '\0') {
    current_data[length] = *data;
    length++;

    if (length == 64) {
      sha256_transform(state, current_data);
      bit_length += 512;
      length = 0;
    }

    data++;
  }

  bit_length += length * 8; // add the remaining length

  // Pad whatever data is left in the buffer.
  if (length < 56) {
    current_data[length++] = 0x80;
    while (length < 56) {
      current_data[length++] = 0x00;
    }
  } else {
    current_data[length++] = 0x80;
    while (length < 64) {
      current_data[length++] = 0x00;
    }
    sha256_transform(state, current_data);
    memset(current_data, 0, 56);
  }

  // Append to the padding the total message's length in bits and transform.
  current_data[63] = bit_length;
  current_data[62] = bit_length >> 8;
  current_data[61] = bit_length >> 16;
  current_data[60] = bit_length >> 24;
  current_data[59] = bit_length >> 32;
  current_data[58] = bit_length >> 40;
  current_data[57] = bit_length >> 48;
  current_data[56] = bit_length >> 56;
  sha256_transform(state, current_data);

  // Since this implementation uses little endian byte ordering and SHA uses big
  // endian, reverse all the bytes when copying the final state to the output
  // hash.
  for (int i = 0; i < 4; i++) {
    result[i] = (state[0] >> (24 - i * 8)) & 0x000000ff;
    result[i + 4] = (state[1] >> (24 - i * 8)) & 0x000000ff;
    result[i + 8] = (state[2] >> (24 - i * 8)) & 0x000000ff;
    result[i + 12] = (state[3] >> (24 - i * 8)) & 0x000000ff;
    result[i + 16] = (state[4] >> (24 - i * 8)) & 0x000000ff;
    result[i + 20] = (state[5] >> (24 - i * 8)) & 0x000000ff;
    result[i + 24] = (state[6] >> (24 - i * 8)) & 0x000000ff;
    result[i + 28] = (state[7] >> (24 - i * 8)) & 0x000000ff;
  }
}

__global__ void sha256_cuda(char *wordlist_cuda, int wordlist_length,
                            char *result_cuda, char *password_cuda) {
  int index = blockIdx.x * 1024 + threadIdx.x;
  char *thread_start = wordlist_cuda + (wordlist_length / 1024 / 1024) * index;
  char *thread_end =
      wordlist_cuda + (wordlist_length / 1024 / 1024) * (index + 1);
  if (thread_end > wordlist_cuda + wordlist_length) {
    thread_end = wordlist_cuda + wordlist_length;
  }

  while (thread_start > wordlist_cuda && *thread_start != '\0') {
    thread_start--;
  }

  while (thread_start < thread_end) {
    char *thread_next = thread_start;
    while (*thread_next != '\0' && *thread_next != '\n') {
      thread_next++;
    }
    *thread_next = '\0';

    char hash[SHA256_LENGTH];
    sha256(thread_start, hash);

    bool is_equal = true;
    for (int i = 0; i < SHA256_LENGTH; i++) {
      if (result_cuda[i] != hash[i]) {
        is_equal = false;
        break;
      }
    }
    if (is_equal) {
      while (*thread_start != '\0') {
        *password_cuda = *thread_start;
        thread_start++;
        password_cuda++;
      }
      return;
    }

    thread_start = thread_next + 1;
  }
}

int main(int argc, char *argv[]) {
  char *wordlist;
  char password[MAX_LINE_LENGTH];

  if (argc != 3) {
    fprintf(stderr, "usage: %s <wordlist> <hash>\n", argv[0]);
    return 1;
  }

  FILE *file = fopen(argv[1], "r");
  assert(file != NULL);

  // Get the size of the file
  fseek(file, 0, SEEK_END);
  long file_size = ftell(file);
  rewind(file);

  wordlist = (char *)malloc(file_size * sizeof(char));
  assert(wordlist != NULL);

  size_t bytes_read = fread(wordlist, sizeof(char), file_size, file);

  fclose(file);
  //   assert(bytes_read == file_size);

  char *wordlist_cuda;
  assert(hipMalloc((void **)&wordlist_cuda, file_size * sizeof(char)) ==
         hipSuccess);
  assert(hipMemcpy(wordlist_cuda, wordlist, file_size * sizeof(char),
                    hipMemcpyHostToDevice) == hipSuccess);

  char result[SHA256_LENGTH];
  load_hex(argv[2], result);
  char *result_cuda;
  assert(hipMalloc((void **)&result_cuda, SHA256_LENGTH) == hipSuccess);
  assert(hipMemcpy(result_cuda, result, SHA256_LENGTH,
                    hipMemcpyHostToDevice) == hipSuccess);

  char *password_cuda;
  assert(hipMalloc((void **)&password_cuda, MAX_LINE_LENGTH * sizeof(char)) ==
         hipSuccess);

  sha256_cuda<<<1024, 1024>>>(wordlist_cuda, file_size, result_cuda,
                              password_cuda);

  assert(hipMemcpy(password, password_cuda, MAX_LINE_LENGTH,
                    hipMemcpyDeviceToHost) == hipSuccess);

  printf("%s\n", password);
}